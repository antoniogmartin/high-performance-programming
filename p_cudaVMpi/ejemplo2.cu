#include "hip/hip_runtime.h"
/**
* Vector addition: C = A + B.
*
* This sample is a very basic sample that implements element by element
* vector addition. It is the same as the sample illustrating Chapter 2
* of the programming guide with some additions like error checking.
*/
#include <time.h>
#include <stdio.h>
#include <cstdlib>
#include <stdlib.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include ""


__global__ void prime_number(int * n, int * p)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j=0;
	int prime=1;

	   if (i < *n)
	    {
		    for ( j = 2; j < i ; j++ )
		    {
		      if ( ( i % j ) == 0 )
		      {
		        prime = 0;
		        break;
		      }
		    }
		    if(i==0||i==1)
		    	prime=0;

		    p[i]=prime;
	    }

}
 

__global__ void reduce( int *a, int *b,int * d_A) {
	
	__shared__  int cache[256];//hebras por bloque
	int tid = threadIdx.x + blockIdx.x * blockDim.x; 
	int cacheIndex = threadIdx.x;
	int temp = 0; 
	int N=*d_A;
	while (tid < N) {
	           temp += a[tid];
	           tid += blockDim.x * gridDim.x;
	}
	    // set the cache values

	     cache[cacheIndex] = temp;

	    // synchronize threads in this block
	__syncthreads();

	// for reductions, threadsPerBlock must be a power of 2 // because of the following code

	int i = blockDim.x/2;
	while (i != 0) {
	if (cacheIndex < i)
	cache[cacheIndex] += cache[cacheIndex + i];
	__syncthreads();
	i /= 2;

	}

	if (cacheIndex == 0) b[blockIdx.x] = cache[0];
}

int main(int argc, char *argv[])
{

	//Mostrar especificaciones de la máquina
	
	clock_t start_cpu, end_cpu;
	double cpu_time_used;
	if (argc != 2){
		printf("1: tama  argc %i", argc);

	}
	else{

		start_cpu = clock();
		/*
		int nDevices;
		hipGetDeviceCount(&nDevices);
		for (int i = 0; i < nDevices; i++) {
			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, i);
			printf("Device Number: %d\n", i);
			printf("  Device name: %s\n", prop.name);
			printf("  Memory Clock Rate (KHz): %d\n",
				prop.memoryClockRate);
			printf("  Memory Bus Width (bits): %d\n",
				prop.memoryBusWidth);
			printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
				2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
		}*/

		// Origen 
		int n=atoi(argv[1]);
		int *h_A = &n;
		

		size_t sizeTama = n * sizeof(int);
		int *h_prime= (int*) malloc(sizeTama);
		int *h_result= (int*) malloc(sizeTama);
		int *h_B= (int*) malloc(sizeTama);
		
		// Destino

		int *d_A = NULL;
		hipMalloc((void **)&d_A, sizeof(int));

		int *d_B = NULL;
		hipMalloc((void **)&d_B,sizeTama);

		int *d_prime=NULL;
		hipMalloc((void **)&d_prime,sizeTama);

		int *d_result=NULL;
		hipMalloc((void **)&d_result,sizeTama);

		//printf("NNA %i\n",*h_A);
		//printf("Copy input data from the host memory to the CUDA device\n");
		
		// Launch the Vector Add CUDA Kernel

		float elapsed = 0, elapsed2=0;

		
		int threadsPerBlock = 256;
		int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
		
		//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

		hipMemcpy(d_A, h_A, sizeof(int), hipMemcpyHostToDevice);

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);

		prime_number << <blocksPerGrid, threadsPerBlock >> >(d_A,d_prime);
		
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);

		 
		hipMemcpy(h_prime, d_prime, sizeTama, hipMemcpyDeviceToHost);

		hipMalloc((void **)&d_prime,sizeTama);
		hipMemcpy(d_prime, h_prime, sizeTama, hipMemcpyHostToDevice);
		hipMemcpy(d_A, h_A, sizeof(int), hipMemcpyHostToDevice);

		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);
		reduce <<<blocksPerGrid, threadsPerBlock >> >(d_prime,d_B,d_A);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed2, start, stop);

		hipMemcpy(h_B, d_B, sizeTama, hipMemcpyDeviceToHost);
		
			
		

		hipEventDestroy(start);
		hipEventDestroy(stop);

		int total_primes=0;

		for (int i=0;i<blocksPerGrid;i++){
			total_primes +=h_B[i];
		}
		end_cpu = clock();
		cpu_time_used = ((double)(end_cpu - start_cpu)/(CLOCKS_PER_SEC));

		printf("%i\t%f \n",n, cpu_time_used + (elapsed*0.001+elapsed2*0.001));

		// Free device global memory
		hipFree(d_A);
		hipFree(d_prime);

		// Free host memory
		//free(h_A);
		free(h_prime);

		hipDeviceReset();
	}


	return 0;
}
