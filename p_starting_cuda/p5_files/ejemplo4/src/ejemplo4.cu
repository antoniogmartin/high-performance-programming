#include "hip/hip_runtime.h"
/**
* Vector addition: C = A + B.
*
* This sample is a very basic sample that implements element by element
* vector addition. It is the same as the sample illustrating Chapter 2
* of the programming guide with some additions like error checking.
*/
#include <ctime>
#include <time.h>
#include <stdio.h>
#include <cstdlib>
#include <stdlib.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include ""
/**
* CUDA Kernel Device code
*
* Computes the vector addition of A and B into C. The 3 vectors have the same
* number of elements numElements.
*/
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	
	float ai, bi, res=0;
	int j=0;
		
	   
	    ai = A[i];

	    bi=B[i];

	    if (i < numElements)
	    {
		for (j=0;j<10000;j++) res+=(ai*bi)/742;
		C[i]=A[i]+B[i];
	
	    }
	

}
//GETLINE FOR NOT-POSIX
__host__ float * readFile(char * path, int * n){

	FILE *  fp;
	fp = fopen(path, "r");
	int lineSize = 1024, pos = 0, pos2 = 0;
	int c;
	int vSize = 10;
	char *buffer = (char *)malloc(lineSize);
	float * v;

	if (fopen == NULL)
		exit(EXIT_FAILURE);

	do { // read all lines in file
		pos = 0;
		do{ // read one line
			c = fgetc(fp);
			if (c != EOF) buffer[pos++] = (char)c;
			if (pos >= lineSize - 1) { // incrementar tamañobuffer - leave room for 0
				lineSize *= 2;
				buffer = (char*)realloc(buffer, lineSize);
			}
		} while (c != EOF && c != '\n');
		buffer[pos] = 0;
		if (pos2 == 0){
			vSize = atoi(buffer);
			v = (float *)malloc(sizeof(float)*vSize);
		}
		else if (pos2 <= vSize){
			v[pos2 - 1] = atof(buffer);
		}
		pos2++;
	} while (c != EOF);

	fclose(fp);
	free(buffer);

	*n = vSize;
	return v;
}

/**
* Host main routine
*/

int main(int argc, char *argv[])
{

	//Mostrar especificaciones de la máquina
	int nDevices;
	clock_t start_cpu, end_cpu;
	double cpu_time_used;
	if (argc != 3){
		printf("1: fichero1, 2: fichero2  agrc %i", argc);

	}
	else{

		start_cpu = clock();
		hipGetDeviceCount(&nDevices);
		for (int i = 0; i < nDevices; i++) {
			hipDeviceProp_t prop;
			hipGetDeviceProperties(&prop, i);
			printf("Device Number: %d\n", i);
			printf("  Device name: %s\n", prop.name);
			printf("  Memory Clock Rate (KHz): %d\n",
				prop.memoryClockRate);
			printf("  Memory Bus Width (bits): %d\n",
				prop.memoryBusWidth);
			printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
				2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
		}

		// Print the vector length to be used, and compute its size




		// Allocate the host input vector A
		float *h_A = NULL;

		// Allocate the host input vector B
		float *h_B = NULL;
		float *h_C = NULL;
		int tama = 0, tama1 = 0;
		h_A = readFile(argv[1], &tama);
		h_B = readFile(argv[2], &tama1);

		printf("TAMA %i\n", tama);
		size_t sizeTama = tama * sizeof(float);
		h_C = (float *)malloc(sizeTama);
		//printf("#[Vector addition of %i %i elements]\n", sizeTama, size);


		// Allocate the device input vector A
		float *d_A = NULL;
		hipMalloc((void **)&d_A, sizeTama);


		// Allocate the device input vector B
		float *d_B = NULL;
		hipMalloc((void **)&d_B, sizeTama);


		// Allocate the device output vector C
		float *d_C = NULL;
		hipMalloc((void **)&d_C, sizeTama);


		// Copy the host input vectors A and B in host memory to the device input vectors in
		// device memory
		//printf("Copy input data from the host memory to the CUDA device\n");
		hipMemcpy(d_A, h_A, sizeTama, hipMemcpyHostToDevice);


		hipMemcpy(d_B, h_B, sizeTama, hipMemcpyHostToDevice);

		end_cpu = clock();
		cpu_time_used = ((double)(end_cpu - start_cpu) / (CLOCKS_PER_SEC));
		// Launch the Vector Add CUDA Kernel

		float elapsed = 0;
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);
		hipEventRecord(start, 0);

		int threadsPerBlock = 256;
		int blocksPerGrid = (tama + threadsPerBlock - 1) / threadsPerBlock;
		//printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
		vectorAdd << <blocksPerGrid, threadsPerBlock >> >(d_A, d_B, d_C, tama);
		hipGetLastError();


		// Copy the device result vector in device memory to the host result vector
		// in host memory.
		hipMemcpy(h_C, d_C, sizeTama, hipMemcpyDeviceToHost);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);

		hipEventElapsedTime(&elapsed, start, stop);

		hipEventDestroy(start);
		hipEventDestroy(stop);
		/*for (int i = 0; i < tama; i++)
			printf("h_A[] %f +h_B[] %f =%f \n", h_A[i], h_B[i], h_C[i]);*/
		printf("%2.10f\n", cpu_time_used + elapsed);

		// Free device global memory
		hipFree(d_A);

		hipFree(d_B);

		hipFree(d_C);

		// Free host memory
		free(h_A);
		free(h_B);
		free(h_C);


		// Reset the device and exit
		// hipDeviceReset causes the driver to clean up all state. While
		// not mandatory in normal operation, it is good practice.  It is also
		// needed to ensure correct operation when the application is being
		// profiled. Calling hipDeviceReset causes all profile data to be
		// flushed before the application exits
		hipDeviceReset();
	}


	return 0;
}
